#include "hip/hip_runtime.h"
#include "kernels.h"
#include <stdio.h>
#include <>

__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

inline __device__ void waitUntilN(TestControl* c, int n) {
    volatile int* signalPtr = &c->s;
    int signalCache = *signalPtr;
    while (signalCache != n) {
        signalCache = *signalPtr;
    }
    printf("[%d, %d] wait %d completed\n", blockIdx.x, threadIdx.x, n);
    char* d = c->buff + threadIdx.x * 8;
    (*(int*)d)++;
}

__global__ void 
waitSignal(TestControl* c) {
    printf("kernel pointer testcontrol %p\n", c);
    printf("enter kernel, %d\n", c->s);
    waitUntilN(c, 2);

    waitUntilN(c, 4);
}


void launchWait(TestControl* c) {
    void* args[1] = {&c};
    // waitSignal<<<dim3(1), dim3(2)>>>(c);
    hipLaunchKernel((void*)waitSignal, dim3(1), dim3(2), args, 0, NULL);
}

template <class T>
__global__ static void SumKernel(T* b1, T* b2, size_t nelem) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= nelem)
    return;
  b1[index] += b2[index];
}
#define BLOCK 512
inline dim3 cuda_gridsize_1d(int n) {
  int x = (n - 1) / BLOCK + 1;
  dim3 d = {(uint)x, 1, 1};
  return d;
}

void sumTwoBufferToFirst(void* b1, void* b2, size_t count, hipStream_t stream) {
  SumKernel<float><<<cuda_gridsize_1d(count), BLOCK, 0, stream>>>(
      (float*)b1, (float*)b2, count);
}

void StreamCreate(hipStream_t *stream){
  int greatest_priority;
  hipError_t err;
  err = hipDeviceGetStreamPriorityRange(NULL, &greatest_priority);
  if (err != hipSuccess) {
    printf("error happend while hipDeviceGetStreamPriorityRange\n");
  }
  err = hipStreamCreateWithPriority(stream, hipStreamNonBlocking, greatest_priority);
  if (err != hipSuccess) {
    printf("error while hipStreamCreateWithPriority\n");
  }
}